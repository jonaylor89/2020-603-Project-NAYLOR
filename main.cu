#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

#include <opencv2/opencv.hpp>
#include <opencv2/core.hpp>

#define THREAD_DIM 16

using namespace cv;
using namespace std;

__global__ void rgbaToGreyscaleGPU(
    uchar4 *rgbaImage, 
    unsigned char *greyImage,
    int rows,
    int cols
)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > cols || y > rows)
    {
        return;
    }

    uchar4 rgba = rgbaImage[y * cols + x];
    unsigned char greyValue =  (0.299f * rgba.x) + (0.587f * rgba.y) + (0.114f * rgba.z);
    printf("%c\n", greyValue);
    greyImage[y * cols + x] = greyValue;
}

/*
__global__ void medianFilterGPU(float* greyImageData, int width, int height, float* filteredImage)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int filterWidth = 3;
    int filterHeight = 3;

    int filter[9] {
        0, 1, 0,
        1, 1, 1,
        0, 1, 0
    };

    int pixelValues[9] {0};

    if (x > cols - width + 1 || y > rows - height + 1)
    {
        return;
    }

    int p = 0;
    for (int hh = 0; hh < filterHeight; hh++) 
    {
        for (int ww = 0; ww < filterWidth; ww++) 
        {
            if (filter[hh * filterWidth + ww] == 1)
            {
                int idx = x * width + y + (hh * filterWidth + ww);
                pixel_value[p] = greyImageData[idx];
                p++;
            }
        }
    }

    // Get median pixel value and assign to filteredImage

}
*/

int readImage(
    std::string filename, 
    uchar4 **inputImage, 
    unsigned char **greyImage,
    int *rows,
    int *cols
)
{

    cv::Mat image;
    cv::Mat imageRGBA;
    cv::Mat imageGrey;

    image = cv::imread(filename.c_str(), IMREAD_COLOR);
    if (image.empty())
    {
        cerr << "Couldn't open file: " << filename << endl;
        return 1;
    }

    cv::cvtColor(image, imageRGBA, COLOR_BGR2RGBA);

    imageGrey.create(image.rows, image.cols, CV_8UC1);

    if (!image.isContinuous() || !imageGrey.isContinuous())
    {
        cerr << "Images aren't continous: " << filename << endl;
        return 1;
    }

    *inputImage = (uchar4 *)imageRGBA.ptr<unsigned char>(0);
    *greyImage = imageGrey.ptr<unsigned char>(0);

    *rows = imageRGBA.rows;
    *cols = imageRGBA.cols;

    return 0;
}

void writeImage(std::string filename, unsigned char *imageGrey)
{
    std::string outFile = "grey_" + filename;
    cv::imwrite(outFile.c_str(), *imageGrey);
}

int main(int argc, char **argv)
{
    if (argc != 2) {
        cerr << "Usage: ./main input_file" << endl;
        exit(1);
    }

    // Define Variables
    int err;

    std::string input_file  = std::string(argv[1]);
    
    int rows;
    int cols;
    int size;
    uchar4 *inputImage;
    unsigned char *inputImageGrey;

    uchar4 *d_rgbaImage;
    unsigned char *d_greyImage;

    struct timespec start, end;

    // Read in image
    err = readImage(input_file, &inputImage, &inputImageGrey, &rows, &cols);
    if (err != 0)
    {
        return 1;
    }

    size = rows * cols;

    // Allocate Memory
    hipMalloc(&d_rgbaImage, sizeof(uchar4) * size);
    hipMalloc(&d_greyImage, sizeof(unsigned char) * size);

    hipMemset(&d_greyImage, 0, sizeof(unsigned char) * size);

    // Copy data to GPU
    hipMemcpy(
        &d_rgbaImage, 
        &inputImage, 
        sizeof(uchar4) * size, 
        hipMemcpyHostToDevice
    );

    clock_gettime(CLOCK_MONOTONIC_RAW, &start);

    // Run kernel(s)
    dim3 blockSize (THREAD_DIM, THREAD_DIM);
    dim3 gridSize (ceil(rows / (float)THREAD_DIM), ceil(cols / (float)THREAD_DIM));
    rgbaToGreyscaleGPU<<< gridSize, blockSize >>>(d_rgbaImage, d_greyImage, rows, cols);

    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    uint64_t diff = (1000000000L * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec) / 1e6;

    printf("[INFO] Greyscale operation lasted %llu ms\n", diff);

    // Copy results to CPU
    hipMemcpy(
        &inputImageGrey, 
        &d_greyImage, 
        sizeof(unsigned char) * size, 
        hipMemcpyDeviceToHost
    );

    // Write Image
    writeImage(input_file, inputImageGrey);

    // Free Memory
    hipFree(&d_rgbaImage);
    hipFree(&d_greyImage);

    return 0;
}
